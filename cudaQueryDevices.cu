


#include <stdio.h>
#include "book.h"


int main(int argc, char **argv) {
	int gpus; 
	hipDeviceProp_t prop;
	HANDLE_ERROR( hipGetDeviceCount(&gpus) );
	printf("you got %d GPUs!\n",gpus);
	int i;
	for(i=0; i<gpus; i++) {
		printf("+-------------------------------------------+\n");
		HANDLE_ERROR( hipGetDeviceProperties(&prop, 0) );
		printf("+ Name: %s                          +\n", prop.name);
		printf("+ Total global memory: %i           +\n", prop.totalGlobalMem);
		printf("+ Shared memory per block: %i            +\n", prop.sharedMemPerBlock);
		printf("+ Warpsize: %i                              +\n", prop.warpSize);
		printf("+ Maximum grid size: %i, %i, %i    +\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("+ Maximum threads per block: %i           +\n", prop.maxThreadsPerBlock);
		printf("+ Total constant memory: %i              +\n", prop.totalConstMem);
		printf("+ Multiprocessorcount: %i (->96 Cores)       +\n", prop.multiProcessorCount);
		printf("+ Compute compatibility: %d.%d                +\n", prop.major, prop.minor);
		printf("+ deviceOverlap: %d                          +\n", prop.deviceOverlap);
	}
	printf("+-------------------------------------------+\n");
	return 0;
}
